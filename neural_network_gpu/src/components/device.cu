/*************************************************************
*   File: device.cu
*
*
*************************************************************/
#include "device.h"

/*************************************************************
 *    STATIC VARIABLES
 *************************************************************/
hipblasHandle_t Device::cublas_hdl;
const int      Device::total_threads;

/*************************************************************
 *    PUBLIC FUNCTIONS
 *************************************************************/
void Device::Device_Create(void)
{
  hipblasCreate(&cublas_hdl);
}

hipblasHandle_t Device::Device_Get_Handle(void)
{
  return cublas_hdl;
}

void Device::Device_Destroy(void)
{
  hipblasDestroy(cublas_hdl);
}
