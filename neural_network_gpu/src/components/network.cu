/*************************************************************
*   File: network.cu
*
*
*************************************************************/
#include <cmath>
#include "network.h"
#include "device.h"
#include "hip/hip_runtime.h"
#include "helper.h"
/*************************************************************
 *    PUBLIC FUNCTIONS
 *************************************************************/
Network::Network(std::vector<std::shared_ptr<Layer>>& group_l, int input_size, int output_size, float lr, int b_size, int epoch):
  layers(group_l),
  in_size(input_size),
  out_size(output_size),
  eta(lr),
  batch_size(b_size),
  epoch_time(epoch)
{
  Helper::cuda_array_zero_allocate(&gpu_input   , Layer::FLOAT_TYPE     , in_size);

#if USING_HALF_FLOAT
  Helper::cuda_array_zero_allocate(&gpu_h_input , Layer::HALF_FLOAT_TYPE, in_size);
  Helper::cuda_array_zero_allocate(&gpu_output  , Layer::FLOAT_TYPE     , out_size);
#endif /* USING_HALF_FLOAT */
}

Network::~Network()
{
  hipFree(gpu_input);

#if USING_HALF_FLOAT
  hipFree(gpu_h_input);
  hipFree(gpu_output);
#endif /* USING_HALF_FLOAT */
}

void Network::Predict(const float * input, float * output)
{

#if USING_HALF_FLOAT
    // Copy to GPU memory
    hipMemcpy(gpu_input, input, sizeof(float) * in_size, hipMemcpyHostToDevice);
    // convert input from float to half float
    Helper::cvtfloat2half(gpu_input, gpu_h_input, in_size);
    // Feed the model
    Layer::layer_param_t network_output = Forward_Propagate(gpu_h_input);
    // convert output from half float to float
    Helper::cvthalf2float(network_output, gpu_output, out_size);
    // Copy back to cpu buffer
    hipMemcpy(output, gpu_output, sizeof(float) * out_size, hipMemcpyDeviceToHost);

#else
    // Copy to GPU memory
    hipMemcpy(gpu_input, input, sizeof(float) * in_size, hipMemcpyHostToDevice);
    // Feed the model
    Layer::layer_param_t network_output = Forward_Propagate(gpu_input);
    // Copy back to cpu buffer
    hipMemcpy(output, network_output, sizeof(float) * out_size, hipMemcpyDeviceToHost);

#endif

}

void Network::Train(const float * input, const float * e_output,  int total_train_samples,
            const float * test_input, const float * test_e_output, int total_test_samples)
{
  std::cout << "Start training....." << std::endl;
  std::cout << "+ Input size    : " << in_size << std::endl;
  std::cout << "+ Output size   : " << out_size << std::endl;
  std::cout << "+ Learning rate : " << eta << std::endl;
  std::cout << "+ Batch size    : " << batch_size << std::endl;
  std::cout << "+ Epoch time    : " << epoch_time << std::endl;
  std::cout << "+ Total samples : " << total_train_samples << std::endl;

#if USING_HALF_FLOAT
  // First, allocate memory in gpu to store:
  //  + a batch input <float>
  //  + a batch input <half float>
  //  + a batch expect output <float>
  //  + a batch expect output <half float>
  //  + a batch neural output <float>
  //  + a batch neural output <half float>
  float *       f_b_input_gpu;
  Layer::layer_param_t hf_b_input_gpu;
  float *       f_b_e_output_gpu;
  Layer::layer_param_t hf_b_e_output_gpu;
  float *       f_b_n_output_gpu;
  Layer::layer_param_t hf_b_n_output_gpu;
  Helper::cuda_array_allocate(&f_b_input_gpu      , Layer::FLOAT_TYPE       , in_size  * batch_size);
  Helper::cuda_array_allocate(&hf_b_input_gpu     , Layer::HALF_FLOAT_TYPE  , in_size  * batch_size);
  Helper::cuda_array_allocate(&f_b_e_output_gpu   , Layer::FLOAT_TYPE       , out_size * batch_size);
  Helper::cuda_array_allocate(&hf_b_e_output_gpu  , Layer::HALF_FLOAT_TYPE  , out_size * batch_size);
  Helper::cuda_array_allocate(&f_b_n_output_gpu   , Layer::FLOAT_TYPE       , out_size * batch_size);
  Helper::cuda_array_allocate(&hf_b_n_output_gpu  , Layer::HALF_FLOAT_TYPE  , out_size * batch_size);

  // Allocate space to store all neural outputs of a single batch on CPU side
  std::unique_ptr<float> cpu_neural_output(new float[out_size * batch_size]);

  // Allocate additional memory for loss derivative
  Layer::layer_param_t loss_dvt;
  Helper::cuda_array_allocate(&loss_dvt           , Layer::HALF_FLOAT_TYPE  , out_size);

  // For each epoch time, do back propagation on training set
  int total_batches = total_train_samples / batch_size;
  for(int ep = 0; ep < epoch_time; ep++)
  {
    for(int b_count = 0; b_count < total_batches; b_count++)
    {
      // In training set, select random batch to do back propagation
      int     b_idx                    = rand() % (total_batches);
      float * b_start_addr             = (float *)input + b_idx * batch_size * in_size;
      float * b_ex_output_start_addr   = (float *)e_output + b_idx * batch_size * out_size;

      // Copy selected batch to gpu memory
      hipMemcpy(f_b_input_gpu    , b_start_addr          , sizeof(float) * in_size  * batch_size, hipMemcpyHostToDevice);
      hipMemcpy(f_b_e_output_gpu , b_ex_output_start_addr, sizeof(float) * out_size * batch_size, hipMemcpyHostToDevice);

      // in gpu memory:
      // + convert input float to half float
      // + convert expect output float to half float
      Helper::cvtfloat2half(f_b_input_gpu   , hf_b_input_gpu    , in_size  * batch_size);
      Helper::cvtfloat2half(f_b_e_output_gpu, hf_b_e_output_gpu , out_size * batch_size);

      // For each training sample in a batch, do back propagation
      for(int s_idx = 0; s_idx < batch_size; s_idx++)
      {
        Layer::layer_param_t sample               = hf_b_input_gpu    + s_idx * in_size;
        Layer::layer_param_t sample_ex_output     = hf_b_e_output_gpu + s_idx * out_size;
        Layer::layer_param_t neural_output;
        // Do forward propagation
        neural_output = Forward_Propagate(sample);
        // Copy to GPU memory pool for neural output
        hipMemcpy(hf_b_n_output_gpu + s_idx * out_size, neural_output, Layer::GPU_DATA_SIZE * out_size, hipMemcpyDeviceToDevice);
        // Calculate loss derivative
        Helper::Cross_Entropy_Loss_Derivative(neural_output, sample_ex_output, loss_dvt, out_size);
        // Back propagate loss derivative through each layer
        Layer::layer_param_t err_signal = loss_dvt;
        for(auto iter = layers.rbegin(); iter != layers.rend(); iter++)
        {
          err_signal = iter->get()->backward_propagation(err_signal);
        }
      }
      // After each batch, update each layer with new weights adn biases
      for(auto l : layers)
      {
        l->update(eta, batch_size);
      }
      // On specific batch index, calculate the loss
      if(b_count % 500 == 0)
      {
        /*************************
         *
         * Calculate loss on training set
         *
         *************************/
        // convert neural outputs from half float to float
        Helper::cvthalf2float(hf_b_n_output_gpu, f_b_n_output_gpu, out_size * batch_size);
        // copy neural outputs to cpu memory since we are going to calculate the loss in cpu side
        hipMemcpy(cpu_neural_output.get(), f_b_n_output_gpu, sizeof(float) * out_size * batch_size, hipMemcpyDeviceToHost);
        // calculate loss
        float batch_loss = 0.0;
        Helper::Cross_Entropy_Loss(cpu_neural_output.get(), b_ex_output_start_addr, &batch_loss, out_size * batch_size);

        std::cout << "\tLoss:  + Train: " << batch_loss << std::endl;
      }
    }
  }

  // Finish training, free GPU memory
  hipFree(f_b_input_gpu);
  hipFree(hf_b_input_gpu);
  hipFree(f_b_e_output_gpu);
  hipFree(hf_b_e_output_gpu);
  hipFree(f_b_n_output_gpu);
  hipFree(hf_b_n_output_gpu);

#else
  // First, allocate memory in gpu to store a batch
  Layer::layer_param_t gpu_b_input;
  Layer::layer_param_t gpu_e_output;
  Layer::layer_param_t gpu_neural_output;
  Layer::layer_param_t loss_dvt;
  Helper::cuda_array_allocate(&gpu_b_input      , Layer::FLOAT_TYPE, in_size  * batch_size);
  Helper::cuda_array_allocate(&gpu_e_output     , Layer::FLOAT_TYPE, out_size * batch_size);
  Helper::cuda_array_allocate(&gpu_neural_output, Layer::FLOAT_TYPE, out_size * batch_size);
  Helper::cuda_array_allocate(&loss_dvt         , Layer::FLOAT_TYPE, out_size);

  // Allocate space to store all neural outputs of a single batch on CPU side
  std::unique_ptr<float> cpu_neural_output(new float[out_size * batch_size]);

  // For each epoch time, do back propagation on training set
  int total_batches = total_train_samples / batch_size;

  for(int ep = 0; ep < epoch_time; ep++)
  {
    for(int b_count = 0; b_count < total_batches; b_count++)
    {
      // In training set, select random batch to do back propagation
      int     b_idx                    = rand() % (total_batches);
      float * b_start_addr             = (float *)input + b_idx * batch_size * in_size;
      float * b_ex_output_start_addr   = (float *)e_output + b_idx * batch_size * out_size;

      // Copy selected batch to gpu memory
      hipMemcpy(gpu_b_input  , b_start_addr          , sizeof(float) * in_size  * batch_size, hipMemcpyHostToDevice);
      hipMemcpy(gpu_e_output , b_ex_output_start_addr, sizeof(float) * out_size * batch_size, hipMemcpyHostToDevice);

      // For each training sample in a batch, do back propagation
      for(int s_idx = 0; s_idx < batch_size; s_idx++)
      {
        Layer::layer_param_t sample        = gpu_b_input   + s_idx * in_size;
        Layer::layer_param_t ex_output     = gpu_e_output  + s_idx * out_size;
        Layer::layer_param_t neural_output;

        // Do forward propagation
        neural_output = Forward_Propagate(sample);

        // Copy to GPU memory pool for neural output
        hipMemcpy(gpu_neural_output + s_idx * out_size, neural_output, sizeof(float) * out_size, hipMemcpyDeviceToDevice);

        // Calculate loss derivative
        Helper::Cross_Entropy_Loss_Derivative(neural_output, ex_output, loss_dvt, out_size);

        // Back propagate loss derivative through each layer
        Layer::layer_param_t err_signal = loss_dvt;
        for(auto iter = layers.rbegin(); iter != layers.rend(); iter++)
        {
          err_signal = iter->get()->backward_propagation(err_signal);
        }

      }

      // After each batch, update each layer with new weights adn biases
      for(auto l : layers)
      {
        l->update(eta, batch_size);
      }

      // On specific batch index, calculate the loss
      if(b_count % 500 == 0)
      {
        /*************************
         *
         * Calculate loss on training set
         *
         *************************/
        float batch_loss = 0.0;
        hipMemcpy(cpu_neural_output.get(), gpu_neural_output, sizeof(float) * out_size * batch_size, hipMemcpyDeviceToHost);
        Helper::Cross_Entropy_Loss(cpu_neural_output.get(), b_ex_output_start_addr, &batch_loss, out_size * batch_size);

        std::cout << "\tLoss:  + Train: " << batch_loss << std::endl;

        // /*************************
        //  *
        //  * Calculate loss on validation set
        //  *
        //  *************************/
        // // select random batch in validation set
        // For test set
        // int total_test_batches = total_test_samples / batch_size;
        // int test_b_idx = rand() % total_test_batches;
        // float * test_b_start_addr             = (float *)test_input + test_b_idx * batch_size * in_size;
        // float * test_b_ex_output_start_addr   = (float *)test_e_output + test_b_idx * batch_size * out_size;
        //
        // // Copy selected batch to gpu memory
        // hipMemcpy(gpu_b_input  , test_b_start_addr          , sizeof(float) * in_size  * batch_size, hipMemcpyHostToDevice);
        // hipMemcpy(gpu_e_output , test_b_ex_output_start_addr, sizeof(float) * out_size * batch_size, hipMemcpyHostToDevice);
        //
        // // For each training sample in a batch, do back propagation
        // for(int s_idx = 0; s_idx < batch_size; s_idx++)
        // {
        //   float * sample        = gpu_b_input   + s_idx * in_size;
        //   float * neural_output;
        //
        //   // Do forward propagation
        //   neural_output = Forward_Propagate(sample);
        //
        //   // Copy to GPU memory pool for neural output
        //   hipMemcpy(gpu_neural_output + s_idx * out_size, neural_output, sizeof(float) * out_size, hipMemcpyDeviceToDevice);
        // }
        //
        // float test_loss = 0.0;
        // hipMemcpy(cpu_neural_output.get(), gpu_neural_output, sizeof(float) * out_size * batch_size, hipMemcpyDeviceToHost);
        // Helper::Cross_Entropy_Loss(cpu_neural_output.get(), test_b_ex_output_start_addr, &test_loss, out_size * batch_size);
        //
        // std::cout << " - Validate: " << test_loss << std::endl;


      }
    }
  }
  // Finish training, free GPU memory
  hipFree(gpu_b_input);
  hipFree(gpu_e_output);
  hipFree(loss_dvt);
  hipFree(gpu_neural_output);

#endif

  std::cout << "Finish training" << std::endl;

}


/*************************************************************
 *  PRIVATE FUNCTIONS
 *************************************************************/
Layer::layer_param_t Network::Forward_Propagate(Layer::layer_param_t input)
{
  Layer::layer_param_t layer_feed = input;

  for(auto l : layers)
  {
    layer_feed = l->forward_propagation(layer_feed);
  }

  return layer_feed;
}
