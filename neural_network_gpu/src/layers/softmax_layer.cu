/*************************************************************
*   File: softmax_layer.cu
*
*
*************************************************************/
#include <iostream>
#include <cmath>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "softmax_layer.h"
#include "../components/device.h"
#include "../components/helper.h"
/*************************************************************
 *    PUBLIC FUNCTIONS
 *************************************************************/
Softmax_Layer::Softmax_Layer(int n_inputs, int n_outputs)
{
  total_inputs  = n_inputs;
  total_outputs = n_outputs;
  Helper::cuda_array_random_allocate( &w, Layer::HALF_FLOAT_TYPE, n_inputs * n_outputs );
  Helper::cuda_array_random_allocate( &b, Layer::HALF_FLOAT_TYPE, n_outputs );

  Helper::cuda_array_zero_allocate( &z        , Layer::HALF_FLOAT_TYPE, n_outputs );
  Helper::cuda_array_zero_allocate( &w_grad   , Layer::HALF_FLOAT_TYPE, n_inputs * n_outputs );
  Helper::cuda_array_zero_allocate( &b_grad   , Layer::HALF_FLOAT_TYPE, n_outputs );
  Helper::cuda_array_zero_allocate( &output   , Layer::HALF_FLOAT_TYPE, n_outputs );
  Helper::cuda_array_zero_allocate( &err      , Layer::HALF_FLOAT_TYPE, n_inputs );
  Helper::cuda_array_zero_allocate( &act_dvt  , Layer::HALF_FLOAT_TYPE, n_outputs );
  Helper::cuda_array_zero_allocate( &err_dvt  , Layer::HALF_FLOAT_TYPE, n_outputs );

}

Softmax_Layer::~Softmax_Layer()
{
  hipFree(w);
  hipFree(b);
  hipFree(z);
  hipFree(w_grad);
  hipFree(b_grad);
  hipFree(output);
  hipFree(err);
  hipFree(act_dvt);
  hipFree(err_dvt);
}

layer_param_t Softmax_Layer::forward_propagation(layer_param_t in)
{
  // Save the input
  input = in;

  // Calculate the net
  // z = w.x + b
  Helper::net_calc(input, w, b, z, total_inputs, total_outputs);

  // Apply Softmax activate function
  // output = softmax(z)
  Helper::softmax_calc(z, output, total_outputs);

  // Return this layer's output for further calculation in next layer
  return output;
}

layer_param_t Softmax_Layer::backward_propagation(layer_param_t error)
{
  // Calculate derivative of neuron output
  // dO/dnet = softmax'(z)
  Helper::softmax_dev_calc(output, act_dvt, total_outputs);

  // Calculate error derivative
  // dE/dnet = dE/dO x dO/dnet
  // dE/dO is error signal from next layer
  Helper::err_dev_calc(error, act_dvt, err_dvt, total_outputs);

  // Accumulate gradients
  // dw = dw + input.dE/dnet
  // db = db + dE/dnet
  Helper::accum_w_grad(input, err_dvt, w_grad, total_inputs, total_outputs);
  Helper::accum_b_grad(err_dvt, b_grad, total_outputs);

  // Calculate error signal propagated to previous layer
  // error_signal = dE/dnet * w
  Helper::err_signal_calc(w, err_dvt, err, total_inputs, total_outputs);

  // Back propagate this layer's error signal
  return err;
}

void Softmax_Layer::update(float eta, int batch_size)
{
  // Update weights and biases and clear gradients
  // w = w - dw * (eta/batch_size)
  // b = b - db * (eta/batch_size)
  float alpha = -eta / batch_size;
  Helper::update_param(w, w_grad, alpha, total_inputs * total_outputs);
  Helper::update_param(b, b_grad, alpha, total_outputs);
}
/*************************************************************
 *    PRIVATE FUNCTIONS
 *************************************************************/
